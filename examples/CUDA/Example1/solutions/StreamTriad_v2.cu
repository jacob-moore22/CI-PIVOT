#include "hip/hip_runtime.h"
/**
 * Stream Triad Benchmark Implementation using CUDA
 * This example demonstrates:
 * 1. Basic CUDA kernel definition and launch
 * 2. Memory management (host and device)
 * 3. Data transfer between host and device
 * 4. Thread indexing and block organization
 * 5. Error handling in CUDA
 * 6. Performance measurement
 */

// CUDA kernel version of Stream Triad
//
// __global__ indicates that this is a kernel function that will be
// executed on the device. This is a special CUDA keyword that tells
// the compiler this function will be called from the host and
// executed on the GPU device. (NOTE: this code must be compiled with
// a CUDA aware compiler such as nvcc or nvc++.)
__global__ void StreamTriad(const int n,        // Total number of elements to process
                            const double scalar,// Scalar multiplier for the triad operation
                            const double *a,    // Input array a
                            const double *b,    // Input array b
                            double *c)          // Output array c
{
  // Get the global thread index to determine which element to process
  // gridDim:   {x,y,z} Number of blocks in the compute grid
  // blockIdx:  {x,y,z} Current block index within the compute grid
  // blockDim:  {x,y,z} Number of threads within each block
  // threadIdx: {x,y,z} Current thread index within the current block
  //
  // NOTE: In this example, we are using a 1D arrays. So, we only need
  // to use the x values
  const int idx = blockIdx.x*blockDim.x+threadIdx.x;
  
  // NOTE: This kernel can be called with any sized grid, with any
  // sized blocks. Often times, the array size is not evenly divisible
  // by the block size. In that case, there may be some allocated
  // threads for which there are no valid array elements. So, it is
  // good practice to add an array bounds check. If the current thread
  // is beyond the array bounds, then do nothing.
  // if (idx >= n) return;

  // TODO: For all remaining threads, perform a single triad operation:
  // e.g.  c[i] = a[i] + scalar*b[i]
  // c[idx] = a[idx] + scalar*b[idx];

  // BONUS: What would happen if this kernel were called with too few
  // blocks and threads? How could you modify this code to handle this
  // condition?
  const int stride = blockDim.x*gridDim.x;    // total number of threads
  
  for (int i=idx; i<n; i+=stride) {
    c[i] = a[i] + scalar*b[i];
  }

}

#include <stdio.h>
#include <sys/time.h>
#include "timer.h"

// Number of times to run the benchmark for averaging
#define NTIMES 16

int main(int argc, char *argv[]){
  // Variables for timing measurements
  struct timespec tkernel, ttotal;  // Structure to hold timing information
  // initializing data and arrays
  int stream_array_size = 80000000;  // Size of arrays (80 million elements)
  double scalar = 3.0, tkernel_sum = 0.0, ttotal_sum = 0.0;  // Scalar value and timing accumulators

  // allocate host memory and initialize
  // Host memory is the CPU's main memory
  double *a = (double *)malloc(stream_array_size*sizeof(double));
  double *b = (double *)malloc(stream_array_size*sizeof(double));
  double *c = (double *)malloc(stream_array_size*sizeof(double));

  // Initialize arrays with values
  // This is done on the CPU before transferring to GPU
  for (int i=0; i<stream_array_size; i++) {
    a[i] = 1.0;
    b[i] = 2.0;
    c[i] = 0.0;  // Initialize c array to 0
  }

  // allocate device memory
  // Device memory is the GPU's memory
  double *a_d, *b_d, *c_d;  // Device pointers (d suffix for device)
  hipError_t err;  // Variable to store CUDA error codes
    
  // Allocate memory on the GPU for each array
  // hipMalloc allocates memory on the GPU device
  err = hipMalloc(&a_d, stream_array_size*sizeof(double));
  if (err != hipSuccess) {
    printf("Error allocating a_d: %s\n", hipGetErrorString(err));
    return -1;
  }
    
  err = hipMalloc(&b_d, stream_array_size*sizeof(double));
  if (err != hipSuccess) {
    printf("Error allocating b_d: %s\n", hipGetErrorString(err));
    return -1;
  }
    
  err = hipMalloc(&c_d, stream_array_size*sizeof(double));
  if (err != hipSuccess) {
    printf("Error allocating c_d: %s\n", hipGetErrorString(err));
    return -1;
  }

  // setting block size and padding total grid size to get even block sizes
  // Block size is the number of threads per block (must be a multiple of 32)
  int blocksize = 512;  // Common block size for modern GPUs
  
  // Calculate grid size to cover all elements, rounding up to ensure
  // full coverage. Restrict grid size so that there are less threads
  // than array elements.
  int gridsize = (stream_array_size + blocksize - 1)/blocksize;

  // Outer benchmark loop
  for (int g=0; (gridsize>>g)>0; ++g) {
    
    // Main benchmark loop
    for (int k = 0; k < NTIMES; k++){
      cpu_timer_start(&ttotal);  // Start total time measurement
        
      // copying array data from host to device
      // hipMemcpyHostToDevice indicates transfer direction
      err = hipMemcpy(a_d, a, stream_array_size*sizeof(double), hipMemcpyHostToDevice);
      if (err != hipSuccess) {
        printf("Error copying a to device: %s\n", hipGetErrorString(err));
        return -1;
      }
        
      err = hipMemcpy(b_d, b, stream_array_size*sizeof(double), hipMemcpyHostToDevice);
      if (err != hipSuccess) {
        printf("Error copying b to device: %s\n", hipGetErrorString(err));
        return -1;
      }
        
      // Synchronize to ensure data transfer is complete
      // This ensures all data is on the GPU before kernel launch
      hipDeviceSynchronize();

      cpu_timer_start(&tkernel);  // Start kernel time measurement
        
      // launch stream triad kernel
      // <<<gridsize, blocksize>>> is CUDA's special syntax for kernel launch configuration
      // This determines how many blocks and threads per block to use
      StreamTriad<<<(gridsize>>2), blocksize>>>(stream_array_size, scalar, a_d, b_d, c_d);
        
      // Check for kernel launch errors
      err = hipGetLastError();
      if (err != hipSuccess) {
        printf("Error launching kernel: %s\n", hipGetErrorString(err));
        return -1;
      }
        
      // Synchronize to ensure kernel completion
      // This ensures the kernel has finished before we try to copy results back
      hipDeviceSynchronize();
      tkernel_sum += cpu_timer_stop(tkernel);  // Record kernel execution time

      // Copy results back from device to host
      // hipMemcpyDeviceToHost indicates transfer direction
      // This operation is blocking, so no need for additional synchronization
      err = hipMemcpy(c, c_d, stream_array_size*sizeof(double), hipMemcpyDeviceToHost);
      if (err != hipSuccess) {
        printf("Error copying c from device: %s\n", hipGetErrorString(err));
        return -1;
      }
        
      ttotal_sum += cpu_timer_stop(ttotal);  // Record total time
        
      // check results and print errors if found. limit to only 10 errors per iteration
      // This verifies the computation was correct
      for (int i=0, icount=0; i<stream_array_size && icount < 10; i++){
        if (fabs(c[i] - (1.0 + 3.0*2.0)) > 1e-10) {  // Use small epsilon for floating point comparison
          printf("Error with result c[%d]=%lf on iter %d\n",i,c[i],k);
          icount++;
        }
      }
    }
    // Print average execution times
    printf("Gridsize: %6d, Average runtime is %lf msecs\n",
           (gridsize>>g), tkernel_sum/NTIMES);
  }

  // free device memory
  // Always free GPU memory when done
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  // free host memory
  // Free CPU memory
  free(a);
  free(b);
  free(c);
    
  return 0;
}